#include <stdio.h>

#include <chrono>
#include <iostream>
#include <vector>
#include <tuple>
#include <mpi.h>
#include <fstream>
#include <ctime>     // Include for time functions
#include <cstdio>
#include <hipsolver.h>

#include "matrix.h"

static int iam;
#define gpuErrchk(ans)                                                         \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "%d: GPUassert: %s %s %d\n", iam, hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

static std::chrono::time_point<std::chrono::system_clock> s,e;
#define START() s = std::chrono::system_clock::now();
#define END() e = std::chrono::system_clock::now();
#define GET()                                                                  \
  (std::chrono::duration_cast<std::chrono::duration<double>>(                   \
       (e = std::chrono::system_clock::now()) - s)                             \
       .count())

using namespace std;
static double *Workspace;
static int Lwork_size = 0;
static const double alpha = 1.0;
static int *status;
static vector<tuple<int,int,int,double>> v_getrf, v_trsm, v_gemm;

void snusolver_LU_gpu(dense_matrix &A, hipsolverHandle_t cusolverHandle) {
  int Lwork;
  int n = A.n, m = A.m;
  if (!n || !m)
    return;
  hipsolverDnDgetrf_bufferSize(cusolverHandle, n, m, A.data_gpu, m, &Lwork);
  if (Lwork > Lwork_size) {
    if (Lwork_size) {
      gpuErrchk(hipFree(Workspace));
    }
    else {
      gpuErrchk(hipMalloc((void **)&status, sizeof(int)));
    }

    gpuErrchk(hipMalloc((void **)&Workspace, Lwork * sizeof(double)));
    Lwork_size = Lwork;
  }

  START();
  hipsolverDnDgetrf(cusolverHandle, n, m, A.data_gpu, m, Workspace, nullptr,
                   status);
  double time = GET();
  static vector<tuple<int,int,int,double>> v_getrf, v_trsm, v_gemm;
  v_getrf.push_back({n,n,n,time});
}
void snusolver_trsm_Lxb_gpu(dense_matrix &L, dense_matrix &b,
                            hipblasHandle_t handle) {
  int n = b.n, m = b.m;
  if (!n || !m)
    return;

  // cblas_dtrsm (CblasRowMajor, CblasLeft, CblasLower, CblasNoTrans,
  // CblasUnit,n,m,1,L.data,n,b.data,m);
  hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
  hipblasOperation_t trans = HIPBLAS_OP_N;
  hipblasDiagType_t diag = HIPBLAS_DIAG_UNIT;

  hipblasDtrsm(handle, side, uplo, trans, diag, n, m, &alpha, L.data_gpu, n,
              b.data_gpu, n);
}
void snusolver_trsm_xUb_gpu(dense_matrix &U, dense_matrix &b,
                            hipblasHandle_t handle) {
  int n = b.n, m = b.m;
  if (!n || !m)
    return;
  // cblas_dtrsm (CblasRowMajor, CblasRight, CblasUpper, CblasNoTrans,
  // CblasNonUnit,n,m,1,U.data,m,b.data,m);

  hipblasSideMode_t side = HIPBLAS_SIDE_RIGHT;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
  hipblasOperation_t trans = HIPBLAS_OP_N;
  hipblasDiagType_t diag = HIPBLAS_DIAG_NON_UNIT;

  hipblasDtrsm(handle, side, uplo, trans, diag, n, m, &alpha, U.data_gpu, m,
              b.data_gpu, n);
  // hipblasDtrsm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N,
  // HIPBLAS_DIAG_NON_UNIT,n,m,&alpha, U.data,m,b.data,n);
}

void snusolver_trsm_Uxb_gpu(dense_matrix &U, dense_matrix &b,
                            hipblasHandle_t handle) {
  int n = b.n, m = b.m;
  if (!n || !m)
    return;
  // cblas_dtrsm (CblasRowMajor, CblasLeft, CblasUpper, CblasNoTrans,
  // CblasNonUnit,n,m,1,U.data,n,b.data,m);
  hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
  hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;
  hipblasOperation_t trans = HIPBLAS_OP_N;
  hipblasDiagType_t diag = HIPBLAS_DIAG_NON_UNIT;

  hipblasDtrsm(handle, side, uplo, trans, diag, n, m, &alpha, U.data_gpu, n,
              b.data_gpu, n);
  // hipblasDtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N,
  // HIPBLAS_DIAG_NON_UNIT,n,m,&alpha, U.data,n,b.data,n);
}

void snusolver_gemm_gpu(dense_matrix &A, dense_matrix &B, dense_matrix &C,
                        hipblasHandle_t handle) {
  int m = A.n, k = A.m, n = B.m;
  // A=m*k, B=k*n, C=m*n
  if (!m || !n || !k)
    return;
  // cblas_dgemm (CblasRowMajor, CblasNoTrans, CblasNoTrans, m,n,k, -1, A.data,
  // k, B.data, n, 1, C.data, n);
  /*hipblasStatus_t hipblasDgemm(hipblasHandle_t handle,
                         hipblasOperation_t transa, hipblasOperation_t transb,
                         int m, int n, int k,
                         const double          *alpha,
                         const double          *A, int lda,
                         const double          *B, int ldb,
                         const double          *beta,
                         double          *C, int ldc)*/
  hipblasOperation_t trans = HIPBLAS_OP_N;
  const double alpha = -1.0;
  const double beta = 1.0;
  hipblasDgemm(handle, trans, trans, m, n, k, &alpha, A.data_gpu, m, B.data_gpu,
              k, &beta, C.data_gpu, m);
}

void flattenData(const vector<tuple<int, int, int, double>>& data, vector<double>& flat) {
    for (const auto& entry : data) {
        int a, b, c;
        double d;
        tie(a, b, c, d) = entry;
        flat.push_back(a);
        flat.push_back(b);
        flat.push_back(c);
        flat.push_back(d);
    }
}

// Function to gather varying data sizes and write to a file in the master process

void gatherAndWriteData() {
    int rank, size;
    MPI_Comm comm=MPI_COMM_WORLD;
    MPI_Comm_rank(comm, &rank);
    MPI_Comm_size(comm, &size);

    // Local data for each process (this can be of varying sizes)
    // Flatten local data into a simple array
    vector<double> flatLocalData;
    flattenData(v_getrf, flatLocalData);

    // Size of local data
    int localCount = flatLocalData.size();

    // Gather the counts of data from each process
    vector<int> recvCounts(size);
    MPI_Gather(&localCount, 1, MPI_INT, recvCounts.data(), 1, MPI_INT, 0, comm);

    // Calculate displacements for receiving data in the master node
    vector<int> displs(size, 0);
    if (rank == 0) {
        for (int i = 1; i < size; ++i) {
            displs[i] = displs[i - 1] + recvCounts[i - 1];
        }
    }

    // Total size of gathered data on the master node
    int totalCount = 0;
    if (rank == 0) {
        totalCount = displs[size - 1] + recvCounts[size - 1];
    }

    // Prepare receive buffer on the master node
    vector<double> gatheredData;
    if (rank == 0) {
        gatheredData.resize(totalCount);
    }

    // Gather varying amounts of data from each process to the master node
    MPI_Gatherv(flatLocalData.data(), localCount, MPI_DOUBLE,
                gatheredData.data(), recvCounts.data(), displs.data(), MPI_DOUBLE,
                0, comm);

    // Master node writes gathered data to a file
    if (rank == 0) {
        // Generate the filename with timestamp
        time_t now = time(NULL);
        struct tm *t = localtime(&now);
        char filename[100];
        snprintf(filename, sizeof(filename), "dense_log_%04d-%02d-%02d_%02d-%02d-%02d.txt",
                 t->tm_year + 1900, t->tm_mon + 1, t->tm_mday,
                 t->tm_hour, t->tm_min, t->tm_sec);

        // Open the file with the generated filename
        ofstream file(filename);
        if (!file.is_open()) {
            cerr << "Unable to open file: " << filename << endl;
            return;
        }

        file << "Gathered Data from all nodes:\n";

        // Write data in chunks based on original process
        for (int i = 0; i < size; ++i) {
            for (int j = 0; j < recvCounts[i]; ++j) {
                file << gatheredData[displs[i] + j] << endl;
            }
        }

        file.close();
        cout << "Data written to file: " << filename << endl;
    }
}