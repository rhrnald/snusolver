#include <algorithm>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <vector>

#include <chrono>
#include <map>
#include <iostream>

#include "kernel.h"
#include "mpi.h"
#include "snusolver.h"

#include "SnuMat.h"

using namespace std;

static int np,iam;
#define gpuErrchk(ans) \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n",
    hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

static hipblasHandle_t handle;
static hipsolverHandle_t cusolverHandle;

void initialize() {
  MPI_Comm_rank(MPI_COMM_WORLD, &iam);
  MPI_Comm_size(MPI_COMM_WORLD, &np);

  char *local_size_env = getenv("OMPI_COMM_WORLD_LOCAL_SIZE");
  int nproc_pernode = local_size_env ? atoi(local_size_env) : -1;
  int ngpu = 4;
  gpuErrchk(hipGetDeviceCount(&ngpu));


  gpuErrchk(hipSetDevice((iam % nproc_pernode) / (nproc_pernode / ngpu)));  //nproc_pernode must be multiple of ngpu

  hipblasCreate(&handle);
  hipsolverDnCreate(&cusolverHandle);
}

void solve(csr_matrix A_csr, double *b, double *x) {
  MPI_Barrier(MPI_COMM_WORLD);
  if(!iam) TIMER_START("Total start");

  SnuMat Ab(A_csr, b, handle, cusolverHandle);
  
  MPI_Barrier(MPI_COMM_WORLD);
  if(!iam) TIMER_START("Factsolve start");
  Ab.solve(x);
  MPI_Barrier(MPI_COMM_WORLD);
  if(!iam) TIMER_END("Factsolve end");

  MPI_Barrier(MPI_COMM_WORLD);
  if(!iam) TIMER_END("Total end");

  gatherAndWriteData();
}